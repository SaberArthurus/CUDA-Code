#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#define THREAD_SIZE 256
using namespace std;


void matgen(float* a, int lda, int n) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			a[i * lda + j] = (float)rand() / RAND_MAX +
				(float)rand() / (RAND_MAX * RAND_MAX);
		}
		//printf("%.2f\n", a[i]);
	}
}

void matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n) {
	int i, j, k;
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			double t = 0;
			for (int k = 0; k < n; ++k) {
				t += a[i * lda + k] * b[k * ldb + j];
			}
			c[i * ldc + j] = t;
		}
	}
}

void compare_mat(const float* a, int lda, const float* b, int ldb, int n) {
	float max_err = 0;
	float average_err = 0;
	for (int i = 0; i < n; ++i){
		for (int j = 0; j < n; ++j) {
			if (b[i * ldb + j] != 0) {
				float err = fabs((a[i * lda + j] - b[i * ldb + j]) / b[i * ldb + j]);
				//printf("%.2f\n", max_err);
				if (max_err < err) max_err = err;
				average_err += err;
			}
		}
	}
	printf("max_err : %f, average_err: %.2f\n", max_err, average_err);
}


__global__ static void matMultCUDA(const float* a, size_t lda,
	const float* b, size_t ldb, float* c, size_t ldc, int n) {
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;
	for (i = tid; i < n; i += blockDim.x) {
		data[i] = a[row * lda + i];
	}
	__syncthreads();
	for (j = tid; j < n; j += blockDim.x) {
		float t = 0;
		float y = 0;
		for (i = 0; i < n; ++i) {
			float r;
			y -= data[i] * b[i * ldb + j];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * ldc + j] = t;
	}
}

clock_t matmultCUDA(const float* a, int lda,
	const float* b, int ldb, float*c, int ldc, int n) {
	float *ac, *bc, *cc;
	clock_t start, end;
	start = clock();
	size_t pitch_a, pitch_b, pitch_c;
	hipMallocPitch((void**)&ac, &pitch_a, sizeof(float)* n, n);
	hipMallocPitch((void**)&bc, &pitch_b, sizeof(float)* n, n);
	hipMallocPitch((void**)&cc, &pitch_c, sizeof(float)* n, n);
	hipMemcpy2D(ac, pitch_a, a, sizeof(float)* lda, sizeof(float)* n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, pitch_b, b, sizeof(float)* ldb, sizeof(float)* n, n, hipMemcpyHostToDevice);

	int blocks = (n + THREAD_SIZE - 1) / THREAD_SIZE;
	matMultCUDA<<<n, THREAD_SIZE, sizeof(float) * n >>>
		(ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float), cc, pitch_c / sizeof(float), n);
	hipMemcpy2D(c, sizeof(float)* n, cc, pitch_c, sizeof(float) * n, n, hipMemcpyDeviceToHost);
	hipFree(ac);
	hipFree(bc);
	hipFree(cc);
	end = clock();
	return end - start;

}
int main(int argc, char** argv) {
	float *a, *b, *c, *d;
	const int n = 1000;
	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);
	srand(10);
	matgen(a, n, n);
	matgen(b, n, n);
	clock_t time = matmultCUDA(a, n, b, n, c, n, n);

	matmult(a, n, b, n, d, n, n);
	compare_mat(c, n, d, n, n);

	double sec = (double)time / CLOCKS_PER_SEC;
	printf("Time used: %.2f  (%.2lf GFLOATS)\n", sec, 2.0 * n * n * n / (sec * 1e9));
	system("pause");
	return 0;
}